#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);                                                                 
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

void readPnm(char * fileName, 
		int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uint8_t * pixels, int width, int height, 
		char * fileName, int numChannels=1)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	if (numChannels == 1)
		fprintf(f, "P2\n");
	else if (numChannels == 3)
		fprintf(f, "P3\n");
	else
	{
		fclose(f);
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height * numChannels; i++)
		fprintf(f, "%hhu\n", pixels[i]);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

char * concatStr(const char * s1, const char * s2)
{
	char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
	strcpy(result, s1);
	strcat(result, s2);
	return result;
}

float computeError(uint8_t * a1, uint8_t * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
		err += abs((int)a1[i] - (int)a2[i]);
	err /= n;
	return err;
}

__global__ void convertRgb2GrayKernel(uchar3 * inPixels, int width, int height, 
		uint8_t * outPixels)
{
	// TODO
    // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;

	if(r < height && c < width)
	{
		int i = r * width + c;
		uint8_t red = inPixels[i].x;
		uint8_t green = inPixels[i].y;
		uint8_t blue = inPixels[i].z;
		outPixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
	}

}

void convertRgb2Gray(uchar3 * inPixels, int width, int height,
		uint8_t * outPixels, 
		bool useDevice=false, dim3 blockSize=dim3(1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
        for (int r = 0; r < height; r++)
        {
            for (int c = 0; c < width; c++)
            {
				int i = r * width + c;
				uint8_t red = inPixels[i].x;
				uint8_t green = inPixels[i].y;
				uint8_t blue = inPixels[i].z;
				outPixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
            }
        }
	}
	else // use device
	{
		// TODO: Allocate device memories
		uchar3 * d_inPixels;
		uint8_t * d_outPixels;
		size_t nBytes = 3 * width * height * sizeof(uint8_t); // 3 phần tử liên tiếp lần lượt là r, g, b nên x3
		CHECK(hipMalloc(&d_inPixels, 3 * width * height * sizeof(uchar3)));
		CHECK(hipMalloc(&d_outPixels, nBytes));

		// TODO: Copy data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, nBytes, hipMemcpyHostToDevice));

		// TODO: Set grid size and call kernel (remember to check kernel error)
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
		convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_outPixels);

		hipError_t errSync  = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess) 
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
		// TODO: Copy result from device memories
		CHECK(hipMemcpy(outPixels, d_outPixels, nBytes, hipMemcpyDeviceToHost));

		// TODO: Free device memories
		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_outPixels));

	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time convertRgb2Gray (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}

__global__ void convertGray2SobelKernel(uint8_t * inPixels, int width, int height, 
		uint8_t * outPixels, int8_t * x_Sobel, int8_t * y_Sobel, uint8_t filterWidth)
{
	
	int outPixelsR = blockIdx.y * blockDim.y + threadIdx.y;
	int outPixelsC = blockIdx.x * blockDim.x + threadIdx.x;

	if(outPixelsR < height && outPixelsC < width)
	{
		int outPixel_x = 0;       
		int outPixel_y = 0;
		for (int filterR = 0; filterR < filterWidth; filterR++)
			{
				for (int filterC = 0; filterC < filterWidth; filterC++)
				{
					int8_t filterVal_x = x_Sobel[filterR * filterWidth + filterC];
					int8_t filterVal_y = y_Sobel[filterR * filterWidth + filterC];
					int inPixelsR = outPixelsR - filterWidth/2 + filterR;
					int inPixelsC = outPixelsC - filterWidth/2 + filterC;
					inPixelsR = min(max(0, inPixelsR), height - 1);
					inPixelsC = min(max(0, inPixelsC), width - 1);
					uint8_t inPixel = inPixels[inPixelsR * width + inPixelsC];
					outPixel_x += filterVal_x * inPixel;
					outPixel_y += filterVal_y * inPixel;
				}
			}
			outPixels[outPixelsR * width + outPixelsC] = abs(outPixel_x) + abs(outPixel_y);
	}

}

void convertGray2Sobel(uint8_t * inPixels, int width, int height,
		uint8_t * outPixels,
		int8_t * x_Sobel, int8_t * y_Sobel, uint8_t filterWidth, bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		for (int outPixelsR = 0; outPixelsR < height; outPixelsR++)
		{
			for (int outPixelsC = 0; outPixelsC < width; outPixelsC++)
			{
				int outPixel_x = 0;       
				int outPixel_y = 0;
				for (int filterR = 0; filterR < filterWidth; filterR++)
				{
					for (int filterC = 0; filterC < filterWidth; filterC++)
					{
						int8_t filterVal_x = x_Sobel[filterR * filterWidth + filterC];
						int8_t filterVal_y = y_Sobel[filterR * filterWidth + filterC];
						int inPixelsR = outPixelsR - filterWidth/2 + filterR;
						int inPixelsC = outPixelsC - filterWidth/2 + filterC;
						inPixelsR = min(max(0, inPixelsR), height - 1);
						inPixelsC = min(max(0, inPixelsC), width - 1);
						uint8_t inPixel = inPixels[inPixelsR * width + inPixelsC];
						outPixel_x += filterVal_x * inPixel;
						outPixel_y += filterVal_y * inPixel;
					}
				}
				outPixels[outPixelsR * width + outPixelsC] = abs(outPixel_x) + abs(outPixel_y);
			}
		}
	}
	else
	{
		// TODO
		uint8_t * d_inPixels;
		uint8_t * d_outPixels;
		int8_t * d_x_Sobel;
		int8_t * d_y_Sobel;
		size_t nBytes = height * width * sizeof(uint8_t);
		size_t nBytesFilter = filterWidth  * filterWidth  * sizeof(int8_t);
		CHECK(hipMalloc(&d_inPixels, nBytes));
		CHECK(hipMalloc(&d_outPixels, nBytes));
		CHECK(hipMalloc(&d_x_Sobel, nBytesFilter));
		CHECK(hipMalloc(&d_y_Sobel, nBytesFilter));
		CHECK(hipMemcpy(d_inPixels, inPixels, nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_x_Sobel, x_Sobel, nBytesFilter, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_y_Sobel, y_Sobel, nBytesFilter, hipMemcpyHostToDevice));
		dim3 gridSize((width - 1) / blockSize.x + 1, 
                (height - 1) / blockSize.y + 1);
		convertGray2SobelKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_outPixels, d_x_Sobel, d_y_Sobel, filterWidth);
		hipError_t errSync  = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess) 
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
		CHECK(hipMemcpy(outPixels, d_outPixels, nBytes, hipMemcpyDeviceToHost));
		CHECK(hipFree(d_inPixels));
        CHECK(hipFree(d_outPixels));
        CHECK(hipFree(d_x_Sobel));
		CHECK(hipFree(d_y_Sobel));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time convertGray2Sobel (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}

//in: inPixels				//out: outPixels, trace
void computeEnergy(uint8_t * inPixels, int width, int height, int * outPixels)
{
	for (int c = 0; c < width; c++)
	{
		outPixels[(height - 1) * width + c] = inPixels[(height - 1) * width + c];
	}
}
__global__ void computeEnergyKernel(uint8_t * inPixels, int width, int height, int * outPixels)
{
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	if (c < width)
	{
		outPixels[(height - 1) * width + c] = inPixels[(height - 1) * width + c];
	}
}

void computeSumEnergy(uint8_t * inPixels, int width, int height,
		int * outPixels, int8_t * trace)
{
	// GpuTimer timer;
	// timer.Start();

	for (int outPixelsR = height - 2; outPixelsR >= 0; outPixelsR--)
	{
		int outPixel_left, outPixel_mid, outPixel_right, temp, temp_sum;
		uint8_t inPixel_cur;
		for (int outPixelsC = 0; outPixelsC < width; outPixelsC++)
		{
			if (outPixelsC == 0)
			{
				inPixel_cur = inPixels[outPixelsR * width];
				outPixel_mid = outPixels[(outPixelsR + 1) * width];
				outPixel_right = outPixels[(outPixelsR + 1) * width + 1];
				if (outPixel_mid < outPixel_right)
				{
					temp = outPixel_mid;
					trace[outPixelsR * width] = 0;
				}
				else
				{
					temp = outPixel_right;
					trace[outPixelsR * width] = 1;
				}
				temp_sum = inPixel_cur + temp;
				outPixels[outPixelsR * width] = temp_sum;
			}
			else if (outPixelsC == width - 1)
			{
				inPixel_cur = inPixels[(outPixelsR + 1) * width - 1];
				outPixel_mid = outPixels[(outPixelsR + 2) * width - 1];
				outPixel_left = outPixels[(outPixelsR + 2) * width - 2];
				if (outPixel_mid < outPixel_left)
				{
					temp = outPixel_mid;
					trace[(outPixelsR + 1) * width - 1] = 0;
				}
				else
				{
					temp = outPixel_left;
					trace[(outPixelsR + 1) * width - 1] = -1;
				}
				temp_sum = inPixel_cur + temp;
				outPixels[(outPixelsR + 1) * width - 1] = temp_sum;
			}
			else
			{
				inPixel_cur = inPixels[outPixelsR * width + outPixelsC];
				outPixel_left = outPixels[(outPixelsR + 1) * width + outPixelsC - 1];
				outPixel_mid = outPixels[(outPixelsR + 1) * width + outPixelsC];
				outPixel_right = outPixels[(outPixelsR + 1) * width + outPixelsC + 1];
				if (outPixel_mid < outPixel_right)
				{
					temp = outPixel_mid;
					trace[outPixelsR * width + outPixelsC] = 0;
				}
				else if (outPixel_left < outPixel_right)
				{
					temp = outPixel_left;
					trace[outPixelsR * width + outPixelsC] = -1;
				}
				else
				{
					temp = outPixel_right;
					trace[outPixelsR * width + outPixelsC] = 1;
				}
				temp_sum = inPixel_cur + temp;
				outPixels[outPixelsR * width + outPixelsC] = temp_sum;
			}
		}
	}
	// timer.Stop();
	// float time = timer.Elapsed();
	// printf("Processing time (SumEnergy): %f ms\n\n", time);
}

__global__ void computeSumEnergyKernel(uint8_t * inPixels, int width, int height, int * outPixels, int8_t * trace)
{
	//
	int outPixelsC = blockIdx.x * blockDim.x + threadIdx.x;
	for (int outPixelsR = height - 2; outPixelsR >= 0; outPixelsR--)
	{
		int outPixel_left, outPixel_mid, outPixel_right, temp, temp_sum;
		uint8_t inPixel_cur;
		if (outPixelsC < width)
		{
			if (outPixelsC == 0)
			{
				inPixel_cur = inPixels[outPixelsR * width];
				outPixel_mid = outPixels[(outPixelsR + 1) * width];
				outPixel_right = outPixels[(outPixelsR + 1) * width + 1];
				if (outPixel_mid < outPixel_right)
				{
					temp = outPixel_mid;
					trace[outPixelsR * width] = 0;
				}
				else
				{
					temp = outPixel_right;
					trace[outPixelsR * width] = 1;
				}
				temp_sum = inPixel_cur + temp;
				outPixels[outPixelsR * width] = temp_sum;
			}
			else if (outPixelsC == width - 1)
			{
				inPixel_cur = inPixels[(outPixelsR + 1) * width - 1];
				outPixel_mid = outPixels[(outPixelsR + 2) * width - 1];
				outPixel_left = outPixels[(outPixelsR + 2) * width - 2];
				if (outPixel_mid < outPixel_left)
				{
					temp = outPixel_mid;
					trace[(outPixelsR + 1) * width - 1] = 0;
				}
				else
				{
					temp = outPixel_left;
					trace[(outPixelsR + 1) * width - 1] = -1;
				}
				temp_sum = inPixel_cur + temp;
				outPixels[(outPixelsR + 1) * width - 1] = temp_sum;
			}
			else
			{
				inPixel_cur = inPixels[outPixelsR * width + outPixelsC];
				outPixel_left = outPixels[(outPixelsR + 1) * width + outPixelsC - 1];
				outPixel_mid = outPixels[(outPixelsR + 1) * width + outPixelsC];
				outPixel_right = outPixels[(outPixelsR + 1) * width + outPixelsC + 1];
				if (outPixel_mid < outPixel_right)
				{
					temp = outPixel_mid;
					trace[outPixelsR * width + outPixelsC] = 0;
				}
				else if (outPixel_left < outPixel_right)
				{
					temp = outPixel_left;
					trace[outPixelsR * width + outPixelsC] = -1;
				}
				else
				{
					temp = outPixel_right;
					trace[outPixelsR * width + outPixelsC] = 1;
				}
				temp_sum = inPixel_cur + temp;
				outPixels[outPixelsR * width + outPixelsC] = temp_sum;
			}
			__syncthreads();
		}
	}
}

void findSeam(int * inPixels, int8_t * trace, int width, int height,
		int * seam)
{
	for (int i = 0; i < height; i++)
	{
		if (i == 0)
		{
			int inPixel_idx = 0;
			for (int c = 1; c < width; c++)
			{
				if (inPixels[c] < inPixels[inPixel_idx])
				{
					inPixel_idx = c;
				}
			}
			seam[i] = inPixel_idx;
		}
		else
		{
			seam[i] = width + seam[i - 1] + trace[seam[i - 1]];
		}
		
	}
}

// __global__ void findSeamKernel(int * inPixels, int8_t * trace, int width, int height, int * seam)
// {
//     int i = blockIdx.y * blockDim.y + threadIdx.y;
// 	if (i < height)
// 	{
// 		if (i == 0)
// 		{
// 			int inPixel_idx = 0;
// 			for (int c = 1; c < width; c++)
// 			{
// 				if (inPixels[c] < inPixels[inPixel_idx])
// 				{
// 					inPixel_idx = c;
// 				}
// 			}
// 			seam[i] = inPixel_idx;
// 		}
// 		else
// 		{
// 			seam[i] = width + seam[i - 1] + trace[seam[i - 1]];
// 		}
// 	}

// }
void removeSeam(uchar3 * inPixels, uint8_t * inPixels_Sobel, int * seam, int width, int height)
{

	int length = width * height;
	for (int i = height - 1; i >= 0; i--)
	{
		int j = 0;
		memcpy(&inPixels_Sobel[seam[i]], &inPixels_Sobel[seam[i] + 1], length - seam[i] - 1 - j);
		memcpy(&inPixels[seam[i]], &inPixels[seam[i] + 1], (length - seam[i] - 1 - j) * sizeof(uchar3));
		j++;
	}
}
__global__ void removeSeamKernel(uchar3 * inPixels, uint8_t * inPixels_Sobel, int * seam, int width, int height)
{
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	if (r < height)
	{
		if (c < width - r - 1)
		{
			int i = seam[r];
			inPixels_Sobel[i] = inPixels_Sobel[i + 1 ];
			inPixels[i] = inPixels[i + 1 ];
		}
		__syncthreads();
	}
}

void find2removeSeam(int new_width, int &i, uint8_t * correctOutSobelPixels, int * correctSumEnergy, int * correctSeam, int8_t * trace, uchar3 * inPixels, int width, int height, bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		for (i; i > new_width; i--)
		{
			computeEnergy(correctOutSobelPixels, i, height, correctSumEnergy);
			
			computeSumEnergy(correctOutSobelPixels, i, height, correctSumEnergy, trace);
			// for (int t=0;t<width;t++)
			// {
			// 	printf("%i, ", correctSumEnergy[(height - 4) * width + t]);
			// }
			findSeam(correctSumEnergy, trace, i, height, correctSeam);
			// for (int t=0;t<height;t++){printf("%i, ", correctSeam[t]);}
			// printf("\n");
			//printf("%i, ", correctSumEnergy[correctSeam[0]]);
			//for (int t=0;t<300;t++){printf("%i, ", correctSumEnergy[t]);}
			removeSeam(inPixels, correctOutSobelPixels, correctSeam, i, height);
			//break;
		}
	}
	else
	{
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
		uchar3 * d_inPixels;
		uint8_t * d_correctOutSobelPixels;
		int * d_correctSumEnergy;
		int * d_correctSeam;
		int8_t * d_trace;
		CHECK(hipMalloc(&d_inPixels, height * width * sizeof(uchar3)));
		CHECK(hipMalloc(&d_correctOutSobelPixels, height * width * sizeof(uint8_t)));
		CHECK(hipMalloc(&d_correctSumEnergy, height * width * sizeof(int)));
		CHECK(hipMalloc(&d_correctSeam, height * sizeof(int)));
		CHECK(hipMalloc(&d_trace, height * width * sizeof(int8_t)));

		CHECK(hipMemcpy(d_correctOutSobelPixels, correctOutSobelPixels, height * width * sizeof(uint8_t), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_inPixels, inPixels, height * width * sizeof(uchar3), hipMemcpyHostToDevice));
		
		dim3 newBlockSize(blockSize.x * blockSize.y);
		dim3 newGridSizeX((width - 1) / newBlockSize.x + 1);
		dim3 newGridSize((width * height - 1) / newBlockSize.x + 1);

		for (i; i > new_width; i--)
		{
			//CHECK(hipMemcpy(d_correctOutSobelPixels, correctOutSobelPixels, height * i * sizeof(uint8_t), hipMemcpyHostToDevice));
			// computeEnergy(correctOutSobelPixels, i, height, correctSumEnergy);
			//computeEnergyKernel<<<gridSize, blockSize>>>(d_correctOutSobelPixels, i, height, d_correctSumEnergy);
			computeEnergyKernel<<<newGridSizeX, newBlockSize>>>(d_correctOutSobelPixels, i, height, d_correctSumEnergy);
			// CHECK(hipMemcpy(correctSumEnergy, d_correctSumEnergy, height * width * sizeof(int), hipMemcpyDeviceToHost));


			// computeSumEnergy(correctOutSobelPixels, i, height, correctSumEnergy, trace);
			//computeSumEnergyKernel<<<gridSize, blockSize>>>(d_correctOutSobelPixels, i, height, d_correctSumEnergy, d_trace);
			computeSumEnergyKernel<<<newGridSizeX, newBlockSize>>>(d_correctOutSobelPixels, i, height, d_correctSumEnergy, d_trace);

			// CHECK(hipMemcpy(correctSumEnergy, d_correctSumEnergy, height * width * sizeof(int), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(trace, d_trace, height * width * sizeof(int8_t), hipMemcpyDeviceToHost));
			
			CHECK(hipMemcpy(trace, d_trace, height * i * sizeof(int8_t), hipMemcpyDeviceToHost));
			CHECK(hipMemcpy(correctSumEnergy, d_correctSumEnergy, height * i * sizeof(int), hipMemcpyDeviceToHost));
			
			// for (int t=0;t<width;t++)
			// {
			// 	printf("%i, ", correctSumEnergy[(height - 4) * width + t]);
			// }
			// CHECK(hipMemcpy(d_correctSumEnergy, correctSumEnergy, height * width * sizeof(int), hipMemcpyHostToDevice));
			// CHECK(hipMemcpy(d_trace, trace, height * width * sizeof(int8_t),hipMemcpyHostToDevice));
			findSeam(correctSumEnergy, trace, i, height, correctSeam);

			// findSeamKernel<<<gridSize, blockSize>>>(d_correctSumEnergy, d_trace, i, height, d_correctSeam);
			// CHECK(hipMemcpy(correctSeam, d_correctSeam, height * sizeof(int), hipMemcpyDeviceToHost));
			// for (int t=0;t<height;t++){printf("%i, ", correctSeam[t]);}
			// printf("\n");
			// printf("%i\n", correctSumEnergy[correctSeam[0]]);
			// for (int t=0;t<300;t++){printf("%i, ", correctSumEnergy[t]);}
			//printf("\ncorrectSeam\n");
			CHECK(hipMemcpy(d_correctSeam, correctSeam, height * sizeof(int), hipMemcpyHostToDevice));
			CHECK(hipMemcpy(d_correctOutSobelPixels, correctOutSobelPixels, height * width * sizeof(uint8_t),hipMemcpyHostToDevice));
			//removeSeam(inPixels, correctOutSobelPixels, correctSeam, i, height);
			removeSeamKernel<<<newGridSize, newBlockSize>>>(d_inPixels, d_correctOutSobelPixels, d_correctSeam, i, height);
			//break;
			
		}
		CHECK(hipMemcpy(inPixels, d_inPixels, height * i * sizeof(uchar3), hipMemcpyDeviceToHost));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time find2removeSeam (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}


int main(int argc, char ** argv)
{	
	// Read input image file
	int width, height;
	uchar3 * inPixels;
	uchar3 * inPixelsDevice;
	readPnm(argv[1], width, height, inPixels);
	readPnm(argv[1], width, height, inPixelsDevice);
	printf("Image size (width x height): %i x %i\n\n", width, height);
	char * outFileNameBase = strtok(argv[2], ".");

	// Set up Sobel filters
	uint8_t filterWidth = 3;
	int8_t * x_Sobel= (int8_t *)malloc(filterWidth * filterWidth);
	int8_t * y_Sobel= (int8_t *)malloc(filterWidth * filterWidth);
	x_Sobel[0] = x_Sobel[6] = y_Sobel[0] = y_Sobel[2] = 1;
	x_Sobel[1] = x_Sobel[4] = x_Sobel[7] = y_Sobel[3] = y_Sobel[4] = y_Sobel[5] = 0;
	x_Sobel[2] = x_Sobel[8] = y_Sobel[6] = y_Sobel[8] = -1;
	x_Sobel[3] = y_Sobel[1] = 2;
	x_Sobel[5] = y_Sobel[7] = -2;

	// Convert RGB to grayscale
	uint8_t * correctOutPixels= (uint8_t *)malloc(width * height);
	convertRgb2Gray(inPixels, width, height, correctOutPixels);
	writePnm(correctOutPixels, width, height, concatStr(outFileNameBase, "_gray_host.pnm"));

	// Convert RGB to grayscale using device
	uint8_t * outPixels= (uint8_t *)malloc(width * height);
	dim3 blockSize(32, 32); // Default
	convertRgb2Gray(inPixelsDevice, width, height, outPixels, true, blockSize);
	writePnm(outPixels, width, height, concatStr(outFileNameBase, "_gray_device.pnm"));

	// Compute mean absolute error between host result and device result
	// float err = computeError(outPixels, correctOutPixels, width * height);
	// printf("Error between device result and host result: %f\n", err);

	// Convert grayscale to sobel-grayscale (energy)
	uint8_t * correctOutSobelPixels= (uint8_t *)malloc(width * height);
	convertGray2Sobel(correctOutPixels, width, height, correctOutSobelPixels, x_Sobel, y_Sobel, filterWidth);
	writePnm(correctOutSobelPixels, width, height, concatStr(outFileNameBase, "_sobel_host.pnm"));

	// Convert grayscale to sobel-grayscale (energy) using device 
	uint8_t * correctOutSobelPixelsDevice= (uint8_t *)malloc(width * height);
	convertGray2Sobel(outPixels, width, height, correctOutSobelPixelsDevice, x_Sobel, y_Sobel, filterWidth, true, blockSize);
	writePnm(correctOutSobelPixelsDevice, width, height, concatStr(outFileNameBase, "_sobel_device.pnm"));

	int new_width = 2 * width / 3; //Default
	int i = width;
	int k = width;

	// Find and remove seam using host
	int * correctSumEnergy = (int *)malloc(width * height * sizeof(int));
	int8_t * trace = (int8_t *)malloc(width * height * sizeof(int8_t));
	int * correctSeam = (int *)malloc(height * sizeof(int));
	find2removeSeam(new_width, i, correctOutSobelPixels, correctSumEnergy, correctSeam, trace, inPixels, width, height);
	writePnm(inPixels, i, height, concatStr(outFileNameBase, "_seam_host.pnm"));

	// Find and remove seam using device
	int * correctSumEnergyDevice = (int *)malloc(width * height * sizeof(int));
	int8_t * traceDevice = (int8_t *)malloc(width * height* sizeof(int8_t));
	int * correctSeamDevice = (int *)malloc(height * sizeof(int));
	find2removeSeam(new_width,k, correctOutSobelPixelsDevice, correctSumEnergyDevice, correctSeamDevice, traceDevice, inPixelsDevice, width, height, true, blockSize);
	writePnm(inPixelsDevice, k, height, concatStr(outFileNameBase, "_seam_device.pnm"));

	// Free memories
	free(inPixels);
	free(outPixels);

}
