#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);                                                                 
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};
void readPnm(char * fileName, 
		int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uint8_t * pixels, int width, int height, 
		char * fileName, int numChannels=1)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	if (numChannels == 1)
		fprintf(f, "P2\n");
	else if (numChannels == 3)
		fprintf(f, "P3\n");
	else
	{
		fclose(f);
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height * numChannels; i++)
		fprintf(f, "%hhu\n", pixels[i]);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

float computeError(uint8_t * a1, uint8_t * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
		err += abs((int)a1[i] - (int)a2[i]);
	err /= n;
	return err;
}

__global__ void convertRgb2GrayKernel(uchar3 * inPixels, int width, int height, 
		uint8_t * outPixels)
{
	// TODO
    // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;

	if(r < height && c < width)
	{
		int i = r * width + c;
		uint8_t red = inPixels[i].x;
		uint8_t green = inPixels[i].y;
		uint8_t blue = inPixels[i].z;
		outPixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
	}

}

void convertRgb2Gray(uchar3 * inPixels, int width, int height,
		uint8_t * outPixels, 
		bool useDevice=false, dim3 blockSize=dim3(1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
        for (int r = 0; r < height; r++)
        {
            for (int c = 0; c < width; c++)
            {
				int i = r * width + c;
				uint8_t red = inPixels[i].x;
				uint8_t green = inPixels[i].y;
				uint8_t blue = inPixels[i].z;
				outPixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
            }
        }
	}
	else // use device
	{
		// TODO: Allocate device memories
		uchar3 * d_inPixels;
		uint8_t * d_outPixels;
		size_t nBytes = 3 * width * height * sizeof(uint8_t); // 3 phần tử liên tiếp lần lượt là r, g, b nên x3
		CHECK(hipMalloc(&d_inPixels, 3 * width * height * sizeof(uchar3)));
		CHECK(hipMalloc(&d_outPixels, nBytes));

		// TODO: Copy data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, nBytes, hipMemcpyHostToDevice));

		// TODO: Set grid size and call kernel (remember to check kernel error)
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
		convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_outPixels);

		hipError_t errSync  = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess) 
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
		// TODO: Copy result from device memories
		CHECK(hipMemcpy(outPixels, d_outPixels, nBytes, hipMemcpyDeviceToHost));

		// TODO: Free device memories
		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_outPixels));

	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time convertRgb2Gray (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}

__global__ void convertGray2SobelKernel(uint8_t * inPixels, int width, int height, 
		uint8_t * outPixels, int8_t * x_Sobel, int8_t * y_Sobel, uint8_t filterWidth)
{
	
	int outPixelsR = blockIdx.y * blockDim.y + threadIdx.y;
	int outPixelsC = blockIdx.x * blockDim.x + threadIdx.x;

	if(outPixelsR < height && outPixelsC < width)
	{
		int outPixel_x = 0;       
		int outPixel_y = 0;
		for (int filterR = 0; filterR < filterWidth; filterR++)
			{
				for (int filterC = 0; filterC < filterWidth; filterC++)
				{
					int8_t filterVal_x = x_Sobel[filterR * filterWidth + filterC];
					int8_t filterVal_y = y_Sobel[filterR * filterWidth + filterC];
					int inPixelsR = outPixelsR - filterWidth/2 + filterR;
					int inPixelsC = outPixelsC - filterWidth/2 + filterC;
					inPixelsR = min(max(0, inPixelsR), height - 1);
					inPixelsC = min(max(0, inPixelsC), width - 1);
					uint8_t inPixel = inPixels[inPixelsR * width + inPixelsC];
					outPixel_x += filterVal_x * inPixel;
					outPixel_y += filterVal_y * inPixel;
				}
			}
			outPixels[outPixelsR * width + outPixelsC] = abs(outPixel_x) + abs(outPixel_y);
	}

}

void convertGray2Sobel(uint8_t * inPixels, int width, int height,
		uint8_t * outPixels,
		int8_t * x_Sobel, int8_t * y_Sobel, uint8_t filterWidth, bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		for (int outPixelsR = 0; outPixelsR < height; outPixelsR++)
		{
			for (int outPixelsC = 0; outPixelsC < width; outPixelsC++)
			{
				int outPixel_x = 0;       
				int outPixel_y = 0;
				for (int filterR = 0; filterR < filterWidth; filterR++)
				{
					for (int filterC = 0; filterC < filterWidth; filterC++)
					{
						int8_t filterVal_x = x_Sobel[filterR * filterWidth + filterC];
						int8_t filterVal_y = y_Sobel[filterR * filterWidth + filterC];
						int inPixelsR = outPixelsR - filterWidth/2 + filterR;
						int inPixelsC = outPixelsC - filterWidth/2 + filterC;
						inPixelsR = min(max(0, inPixelsR), height - 1);
						inPixelsC = min(max(0, inPixelsC), width - 1);
						uint8_t inPixel = inPixels[inPixelsR * width + inPixelsC];
						outPixel_x += filterVal_x * inPixel;
						outPixel_y += filterVal_y * inPixel;
					}
				}
				outPixels[outPixelsR * width + outPixelsC] = abs(outPixel_x) + abs(outPixel_y);
			}
		}
	}
	else
	{
		// TODO
		uint8_t * d_inPixels;
		uint8_t * d_outPixels;
		int8_t * d_x_Sobel;
		int8_t * d_y_Sobel;
		size_t nBytes = height * width * sizeof(uint8_t);
		size_t nBytesFilter = height * width * sizeof(int8_t);
		CHECK(hipMalloc(&d_inPixels, nBytes));
		CHECK(hipMalloc(&d_outPixels, nBytes));
		CHECK(hipMalloc(&d_x_Sobel, nBytesFilter));
		CHECK(hipMalloc(&d_y_Sobel, nBytesFilter));
		CHECK(hipMemcpy(d_inPixels, inPixels, nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_x_Sobel, x_Sobel, nBytesFilter, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_y_Sobel, y_Sobel, nBytesFilter, hipMemcpyHostToDevice));
		dim3 gridSize((width - 1) / blockSize.x + 1, 
                (height - 1) / blockSize.y + 1);
		convertGray2SobelKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_outPixels, d_x_Sobel, d_y_Sobel, filterWidth);
		hipError_t errSync  = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess) 
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
		CHECK(hipMemcpy(outPixels, d_outPixels, nBytes, hipMemcpyDeviceToHost));
		CHECK(hipFree(d_inPixels));
        CHECK(hipFree(d_outPixels));
        CHECK(hipFree(d_x_Sobel));
		CHECK(hipFree(d_y_Sobel));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time convertGray2Sobel (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}
char * concatStr(const char * s1, const char * s2)
{
	char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
	strcpy(result, s1);
	strcat(result, s2);
	return result;
}

int main(int argc, char ** argv)
{	
	// if (argc != 3 && argc != 5)
	// {
	// 	printf("The number of arguments is invalid\n");
	// 	return EXIT_FAILURE;
	// }

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("Image size (width x height): %i x %i\n\n", width, height);
	char * outFileNameBase = strtok(argv[2], ".");

	// Set up Sobel filters
	uint8_t filterWidth = 3;
	int8_t * x_Sobel= (int8_t *)malloc(filterWidth * filterWidth);
	int8_t * y_Sobel= (int8_t *)malloc(filterWidth * filterWidth);
	x_Sobel[0] = x_Sobel[6] = y_Sobel[0] = y_Sobel[2] = 1;
	x_Sobel[1] = x_Sobel[4] = x_Sobel[7] = y_Sobel[3] = y_Sobel[4] = y_Sobel[5] = 0;
	x_Sobel[2] = x_Sobel[8] = y_Sobel[6] = y_Sobel[8] = -1;
	x_Sobel[3] = y_Sobel[1] = 2;
	x_Sobel[5] = y_Sobel[7] = -2;

	// Convert RGB to grayscale
	uint8_t * correctOutPixels= (uint8_t *)malloc(width * height);
	convertRgb2Gray(inPixels, width, height, correctOutPixels);
	writePnm(correctOutPixels, width, height, concatStr(outFileNameBase, "_gray_host.pnm"));

	// Convert RGB to grayscale using device
	uint8_t * outPixels= (uint8_t *)malloc(width * height);
	dim3 blockSize(32, 32); // Default
	convertRgb2Gray(inPixels, width, height, outPixels, true, blockSize);
	writePnm(outPixels, width, height, concatStr(outFileNameBase, "_gray_device.pnm"));

	// Compute mean absolute error between host result and device result
	// float err = computeError(outPixels, correctOutPixels, width * height);
	// printf("Error between device result and host result: %f\n", err);


	// Convert grayscale to sobel-grayscale (energy)
	uint8_t * correctOutSobelPixels= (uint8_t *)malloc(width * height);
	convertGray2Sobel(correctOutPixels, width, height, correctOutSobelPixels, x_Sobel, y_Sobel, filterWidth);
	writePnm(correctOutSobelPixels, width, height, concatStr(outFileNameBase, "_sobel_host.pnm"));

	// Convert grayscale to sobel-grayscale (energy) using device 
	uint8_t * correctOutSobelPixelsDevice= (uint8_t *)malloc(width * height);
	convertGray2Sobel(correctOutPixels, width, height, correctOutSobelPixelsDevice, x_Sobel, y_Sobel, filterWidth, true, blockSize);
	writePnm(correctOutSobelPixelsDevice, width, height, concatStr(outFileNameBase, "_sobel_device.pnm"));


	// Free memories
	free(inPixels);
	free(outPixels);
}
