#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);                                                                 
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

void readPnm(char * fileName, 
		int &numChannels, int &width, int &height, uint8_t * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	if (strcmp(type, "P2") == 0)
		numChannels = 1;
	else if (strcmp(type, "P3") == 0)
		numChannels = 3;
	else // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);

	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uint8_t *)malloc(width * height * numChannels);
	for (int i = 0; i < width * height * numChannels; i++)
		fscanf(f, "%hhu", &pixels[i]);

	fclose(f);
}

void writePnm(uint8_t * pixels, int numChannels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	if (numChannels == 1)
		fprintf(f, "P2\n");
	else if (numChannels == 3)
		fprintf(f, "P3\n");
	else
	{
		fclose(f);
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height * numChannels; i++)
		fprintf(f, "%hhu\n", pixels[i]);

	fclose(f);
}

__global__ void convertRgb2GrayKernel(uint8_t * inPixels, int width, int height, 
		uint8_t * outPixels)
{
	// TODO
    // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;

	if(r < height && c < width)
	{
		int i = r * width + c;
		uint8_t red = inPixels[3 * i];
		uint8_t green = inPixels[3 * i + 1];
		uint8_t blue = inPixels[3 * i + 2];
		outPixels[i] = 0.299*red + 0.587*green + 0.114*blue;  
	}

}

void convertRgb2Gray(uint8_t * inPixels, int width, int height,
		uint8_t * outPixels, 
		bool useDevice=false, dim3 blockSize=dim3(1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
        // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  
        for (int r = 0; r < height; r++)
        {
            for (int c = 0; c < width; c++)
            {
                int i = r * width + c;
                uint8_t red = inPixels[3 * i];
                uint8_t green = inPixels[3 * i + 1];
                uint8_t blue = inPixels[3 * i + 2];
                outPixels[i] = 0.299f*red + 0.587f*green + 0.114f*blue;
            }
        }
	}
	else // use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO: Allocate device memories
		uint8_t * d_inPixels;
		uint8_t * d_outPixels;
		size_t nBytes = 3 * width * height * sizeof(uint8_t); // 3 phần tử liên tiếp lần lượt là r, g, b nên x3
		CHECK(hipMalloc(&d_inPixels, nBytes));
		CHECK(hipMalloc(&d_outPixels, nBytes));

		// TODO: Copy data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, nBytes, hipMemcpyHostToDevice));

		// TODO: Set grid size and call kernel (remember to check kernel error)
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
		convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_outPixels);

		hipError_t errSync  = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess) 
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
		// TODO: Copy result from device memories
		CHECK(hipMemcpy(outPixels, d_outPixels, nBytes, hipMemcpyDeviceToHost));

		// TODO: Free device memories
		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_outPixels));

	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}

__global__ void convertGray2SobelKernel(uint8_t * inPixels, int width, int height, 
		uint8_t * d_outPixels, int8_t * x_Sobel, int8_t * y_Sobel, uint8_t filterWidth)
{
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;

	int radius = filterWidth / 2;

	if(r < height && c < width)
	{
		int i = r * width + c;
		int outPixel_x = 0;
		int outPixel_y = 0;

		for(int filterR = 0; filterR < filterWidth; filterR++)
		{
			for(int filterC = 0; filterC < filterWidth; filterC++)
			{
				int filter_idx = filterR * filterWidth + filterC;
				int8_t filterVal_x = x_Sobel[filter_idx]; 
				int8_t filterVal_y = y_Sobel[filter_idx];
				int inPixelsR = r - radius + filterR;
				int inPixelsC = c - radius + filterC;
				inPixelsR = min(max(0, inPixelsR), height - 1);
				inPixelsC = min(max(0, inPixelsC), width - 1);
				uint8_t inPixel = inPixels[inPixelsR * width + inPixelsC];
				outPixel_x += filterVal_x * inPixel;
				outPixel_y += filterVal_y * inPixel;
			}
		}
		d_outPixels[i] = abs(outPixel_x) + abs(outPixel_y);
	}
}

void convertGray2Sobel(uint8_t * inPixels, int width, int height,
		uint8_t * outPixels, int8_t * x_Sobel, int8_t * y_Sobel, 
		uint8_t filterWidth, bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer2;
	timer2.Start();
	if (useDevice == false)
	{
		for (int outPixelsR = 0; outPixelsR < height; outPixelsR++)
		{
			for (int outPixelsC = 0; outPixelsC < width; outPixelsC++)
			{
				int outPixel_x = 0;       
				int outPixel_y = 0;
				for (int filterR = 0; filterR < filterWidth; filterR++)
				{
					for (int filterC = 0; filterC < filterWidth; filterC++)
					{
						int8_t filterVal_x = x_Sobel[filterR * filterWidth + filterC];
						int8_t filterVal_y = y_Sobel[filterR * filterWidth + filterC];
						int inPixelsR = outPixelsR - filterWidth/2 + filterR;
						int inPixelsC = outPixelsC - filterWidth/2 + filterC;
						inPixelsR = min(max(0, inPixelsR), height - 1);
						inPixelsC = min(max(0, inPixelsC), width - 1);
						uint8_t inPixel = inPixels[inPixelsR * width + inPixelsC];
						outPixel_x += filterVal_x * inPixel;
						outPixel_y += filterVal_y * inPixel;
					}
				}
				outPixels[outPixelsR * width + outPixelsC] = abs(outPixel_x) + abs(outPixel_y);
			}
		}
	}
	else
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		uint8_t * d_inPixels;
		uint8_t * d_outPixels;
		int8_t * d_x_Sobel;
		int8_t * d_y_Sobel;
		size_t nBytes = width * height * sizeof(uint8_t);
		size_t filter_nBytes = filterWidth * filterWidth * sizeof(int8_t);

		CHECK(hipMalloc(&d_inPixels, nBytes));
		CHECK(hipMalloc(&d_outPixels, nBytes));
		CHECK(hipMalloc(&d_x_Sobel, filter_nBytes));
		CHECK(hipMalloc(&d_y_Sobel, filter_nBytes));

		CHECK(hipMemcpy(d_inPixels, inPixels, nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_x_Sobel, x_Sobel, filter_nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_y_Sobel, y_Sobel, filter_nBytes, hipMemcpyHostToDevice));

		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
		convertGray2SobelKernel<<<gridSize,blockSize>>>(d_inPixels, width, height, d_outPixels, d_x_Sobel, d_y_Sobel, filterWidth);

		hipError_t errSync  = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess) 
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
		
		CHECK(hipMemcpy(outPixels, d_outPixels, nBytes, hipMemcpyDeviceToHost));

		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_outPixels));
		CHECK(hipFree(d_x_Sobel));
		CHECK(hipFree(d_y_Sobel));
	}
	timer2.Stop();
	float res = timer2.Elapsed();
	printf("Processing time (%s): %f ms\n", 
    		useDevice == true? "use device" : "use host", res);
}

float computeError(uint8_t * a1, uint8_t * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
		err += abs((int)a1[i] - (int)a2[i]);
	err /= n;
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
	char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
	strcpy(result, s1);
	strcat(result, s2);
	return result;
}

int main(int argc, char ** argv)
{	
	if (argc != 3 && argc != 5)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input RGB image file
	int numChannels, width, height;
	uint8_t * inPixels;
	readPnm(argv[1], numChannels, width, height, inPixels);
	if (numChannels != 3)
		return EXIT_FAILURE; // Input image must be RGB
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Convert RGB to grayscale not using device
	uint8_t * correctOutPixels= (uint8_t *)malloc(width * height);
	convertRgb2Gray(inPixels, width, height, correctOutPixels);

	// Convert RGB to grayscale using device
	uint8_t * outPixels= (uint8_t *)malloc(width * height);
	dim3 blockSize(32, 32); // Default
	if (argc == 5)
	{
		blockSize.x = atoi(argv[3]);
		blockSize.y = atoi(argv[4]);
	} 
	convertRgb2Gray(inPixels, width, height, outPixels, true, blockSize); 

	// Compute mean absolute error between host result and device result
	float err = computeError(outPixels, correctOutPixels, width * height);
	printf("Error between device result and host result: %f\n", err);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(correctOutPixels, 1, width, height, concatStr(outFileNameBase, "_gray_host.pnm"));
	writePnm(outPixels, 1, width, height, concatStr(outFileNameBase, "_gray_device.pnm"));

	// Set up Sobel filters
	uint8_t filterWidth = 3;
	int8_t * x_Sobel= (int8_t *)malloc(filterWidth * filterWidth);
	int8_t * y_Sobel= (int8_t *)malloc(filterWidth * filterWidth);
	x_Sobel[0] = x_Sobel[6] = y_Sobel[0] = y_Sobel[2] = 1;
	x_Sobel[1] = x_Sobel[4] = x_Sobel[7] = y_Sobel[3] = y_Sobel[4] = y_Sobel[5] = 0;
	x_Sobel[2] = x_Sobel[8] = y_Sobel[6] = y_Sobel[8] = -1;
	x_Sobel[3] = y_Sobel[1] = 2;
	x_Sobel[5] = y_Sobel[7] = -2;

	// Convert grayscale to sobel-grayscale not using device
	uint8_t * OutSobelPixels= (uint8_t *)malloc(width * height);
	uint8_t * correctOutSobelPixels= (uint8_t *)malloc(width * height);
	convertGray2Sobel(outPixels, width, height, correctOutSobelPixels, x_Sobel, y_Sobel, filterWidth);
	writePnm(correctOutSobelPixels, 1, width, height, concatStr(outFileNameBase, "_sobel_host.pnm"));

	convertGray2Sobel(outPixels, width, height, OutSobelPixels, x_Sobel, y_Sobel, filterWidth, true, blockSize);
	writePnm(OutSobelPixels, 1, width, height, concatStr(outFileNameBase, "_sobel_device.pnm"));

	// Free memories
	free(inPixels);
	free(outPixels);
}
